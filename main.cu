/*
code example 
written by Daniel Cazacu
this code comes with no warranty, use at your own risk
asd
looooplooooper@gmail.com
*/




#include "hip/hip_runtime.h"

#include <ctime>
#include <cstdio>
#include <cmath>


__global__ void primes_in_range(int *result)
{

	double number = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (number >= 10000000)
	{
		return;
	}

	if (fmod(number,2.0) == 0) return;
	double c=sqrt(number);
	for (double divisor = 3; divisor < c; divisor += 2)
	{
		if (fmod(number,divisor) == 0)
		{
			return;
		}
	}
	
	printf("%f \n", number);
	
	atomicAdd(result, 1);
}

int main()
{
	auto begin = std::clock();

	int *result;
	hipMallocManaged(&result, 4);
	*result = 0;

	primes_in_range<<<10000, 1024>>>(result);
	hipDeviceSynchronize();

	auto end = std::clock();
	auto duration = double(end - begin) / CLOCKS_PER_SEC * 1000;
	
	printf("%d prime numbers found in %d milliseconds", 
		*result, 
		static_cast<int>(duration)
	);
	
	getchar();
	return 0;
}
